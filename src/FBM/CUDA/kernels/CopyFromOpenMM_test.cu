#include "hip/hip_runtime.h"
#include "CopyFromOpenMM_kernel.cu"

extern "C" void TestCopyFrom( const int n, float4* input, float* output) {
    float* in_gpu_positions;
    hipMalloc( (void**) &in_gpu_positions, n * sizeof(float4) );
    hipMemcpy(in_gpu_positions, input, n * sizeof(float4), hipMemcpyHostToDevice );

    float* out_gpu_positions;
    hipMalloc( (void**) &out_gpu_positions, n * 3 * sizeof(float) );
    hipMemcpy(out_gpu_positions, output, n * 3 * sizeof(float), hipMemcpyHostToDevice );

    copyFromOpenMM<<<3 * n, 1>>>(out_gpu_positions, in_gpu_positions, n * 3);

    hipMemcpy(output, out_gpu_positions, n * 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(in_gpu_positions);
    hipFree(out_gpu_positions);
}
