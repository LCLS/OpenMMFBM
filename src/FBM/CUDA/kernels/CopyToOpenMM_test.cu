#include "hip/hip_runtime.h"
#include "CopyToOpenMM_kernel.cu"

extern "C" void TestCopyTo( const int n, float* input, float4* output ) {
    float *out_gpu_positions;
    hipMalloc( ( void ** ) &out_gpu_positions, n * sizeof( float4 ) );
    hipMemcpy( out_gpu_positions, output, n * sizeof( float4 ), hipMemcpyHostToDevice );

    float *in_gpu_positions;
    hipMalloc( ( void ** ) &in_gpu_positions, 3 * n * sizeof( float ) );
    hipMemcpy( in_gpu_positions, input, 3 * n * sizeof( float ), hipMemcpyHostToDevice );

    copyToOpenMM<<<3 * n, 1>>>( out_gpu_positions, in_gpu_positions, 3 * n );

    hipMemcpy( output, out_gpu_positions, n * sizeof( float4 ), hipMemcpyDeviceToHost );

    hipFree( in_gpu_positions );
    hipFree( out_gpu_positions );
}
