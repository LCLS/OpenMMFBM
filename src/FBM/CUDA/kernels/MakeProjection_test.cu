#include "MakeProjection_kernel.cu"

#define THREADSPERBLOCK 512

extern "C" void TestMakeProjection( int n, int m, float* eigvec, int* indices, float* E, float* Et ){
    float *gpuEt;
    hipMalloc( ( void ** ) &gpuEt, m * n * sizeof( float ) );

    float *gpuE;
    hipMalloc( ( void ** ) &gpuE, n * m * sizeof( float ) );

    float *gpuEigvec;
    hipMalloc( ( void ** ) &gpuEigvec, n * m * sizeof( float ) );
    hipMemcpy( gpuEigvec, eigvec, n * m * sizeof( float ), hipMemcpyHostToDevice );

    int *gpuIndices;
    hipMalloc( ( void ** ) &gpuIndices, m * sizeof( int ) );
    hipMemcpy( gpuIndices, indices, m * sizeof( int ), hipMemcpyHostToDevice );

    if( m * n <= THREADSPERBLOCK ) {
        makeProjection <<< 1, m*n >>>( gpuEt, gpuE, gpuEigvec, gpuIndices, m, n );
    }else{
        makeProjection <<< ( m * n ) / THREADSPERBLOCK + 1, THREADSPERBLOCK >>>( gpuEt, gpuE, gpuEigvec, gpuIndices, m, n );
    }

    hipMemcpy( Et, gpuEt, m * n * sizeof( float ), hipMemcpyDeviceToHost );
    hipMemcpy( E, gpuE, m * n * sizeof( float ), hipMemcpyDeviceToHost );

    hipFree( gpuEt );
    hipFree( gpuE );
    hipFree( gpuEigvec );
    hipFree( gpuIndices );
}
