#include "hip/hip_runtime.h"
#include "OddEvenSort_kernel.cu"

#define THREADSPERBLOCK 512

extern "C" void TestOddEvenSort( const int n, float* eigenvalues, float* eigenvectors ) {
    float* gpu_eigenvalues;
    hipMalloc( (void**) &gpu_eigenvalues, n*sizeof(float));
    hipMemcpy( gpu_eigenvalues, eigenvalues, n*sizeof(float), hipMemcpyHostToDevice);

    float* gpu_eigenvectors;
    hipMalloc( (void**) &gpu_eigenvectors, n*n*sizeof(float));
    hipMemcpy( gpu_eigenvectors, eigenvectors, n*n*sizeof(float), hipMemcpyHostToDevice);

    int oddcount = n/2;
    int evencount = (n % 2 == 0) ? oddcount : oddcount+1;

    for (int i = 0; i < ceil(n/2); i++) {
        if (oddcount <= THREADSPERBLOCK) {
            oddEvenEigSort<<<1, oddcount>>>(gpu_eigenvalues, gpu_eigenvectors, n);
        } else {
            oddEvenEigSort<<<oddcount/THREADSPERBLOCK + 1, THREADSPERBLOCK>>>(gpu_eigenvalues, gpu_eigenvectors, n);
        }

        if (evencount <= THREADSPERBLOCK) {
            oddEvenEigSort<<<1, evencount>>>(gpu_eigenvalues, gpu_eigenvectors, n, 1);
        } else {
            oddEvenEigSort<<<evencount/THREADSPERBLOCK + 1, THREADSPERBLOCK>>>(gpu_eigenvalues, gpu_eigenvectors, n);
        }
    }

    hipMemcpy(eigenvalues, gpu_eigenvalues, n*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(eigenvectors, gpu_eigenvectors, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpu_eigenvalues);
    hipFree(gpu_eigenvectors);
}
