#include "Symmetrize1D_kernel.cu"

#define THREADSPERBLOCK 512

extern "C" void TestSymmetrize1D( const size_t blocks, const size_t totalSize, float* output, float* blockHessian, float* blockSizes, int* startDof ) {
    float *gpuBlockHessian;
    hipMalloc( ( void ** ) &gpuBlockHessian, totalSize * sizeof( float ) );
    hipMemcpy( gpuBlockHessian, blockHessian, totalSize * sizeof( float ), hipMemcpyHostToDevice );

    int *gpuBlockSizes;
    hipMalloc( ( void ** ) &gpuBlockSizes, blocks * sizeof( int ) );
    hipMemcpy( gpuBlockSizes, blockSizes, blocks * sizeof( int ), hipMemcpyHostToDevice );

    int *gpuBlockPositions;
    hipMalloc( ( void ** ) &gpuBlockPositions, blocks * sizeof( int ) );
    hipMemcpy( gpuBlockPositions, startDof, blocks * sizeof( int ),	hipMemcpyHostToDevice );

    symmetrize1D <<< blocks / (THREADSPERBLOCK + 1), THREADSPERBLOCK>>>( gpuBlockHessian, gpuBlockPositions, gpuBlockSizes, blocks );

    float outBlockHessian[totalSize];
    hipMemcpy( outBlockHessian, gpuBlockHessian, totalSize * sizeof( float ), hipMemcpyDeviceToHost );

    hipFree( gpuBlockHessian );
    hipFree( gpuBlockSizes );
    hipFree( gpuBlockPositions );
}
