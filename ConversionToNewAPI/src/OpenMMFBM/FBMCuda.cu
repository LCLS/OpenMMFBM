#include "hip/hip_runtime.h"
#include "OpenMMFBM/FBMCuda.h"
#include "kernels/Kernels.cu"
#include "kernels/qr.cu"
//#include <hip/hip_runtime.h>

#include "cula_lapack_device.h"
#include "cula.h"

#include "openmm/internal/ContextImpl.h"
#include "gputypes.h"
#include "OpenMM.h"
using namespace OpenMM;

#include <iostream>
using namespace std;

#define MAXTHREADSPERBLOCK 512
#define CONSERVEDDEGREESOFFREEDOM 6

void FBMCuda::getBlockHessian(std::vector<std::vector<double > >& blockHessianVectors) const {
  blockHessianVectors.resize(_3N, std::vector<double>(_3N, 0.0));

  int size = 0;
  int enddof;
  for(int i = 0; i < numHessBlocks; i++) {
    const int startdof = blocknums[i];
    if (i == numHessBlocks-1) enddof = _3N;
    else enddof = blocknums[i+1];
    size += (enddof - startdof) * (enddof - startdof);
  }

  vector<float> buffer(size, 0.0f);

  hipMemcpy(&buffer[0], blockHessian, size, hipMemcpyDeviceToHost);

  int pos = 0;
  for (int i = 0; i < numHessBlocks; i++) {
    const int startdof = blocknums[i];
    if (i == numHessBlocks-1) enddof = _3N;
    else enddof = blocknums[i+1];
    for (int j = startdof; j < enddof; j++) {
      for (int k = startdof; k < enddof; k++) {
        blockHessianVectors[j][k] = buffer[pos];
        pos++;
      }
    }
  }
}

void FBMCuda::getBlockEigenvectors(std::vector<std::vector<double > >& blockVectors) const {
  blockVectors.resize(_3N, std::vector<double>(_3N, 0.0));
  vector<float> buffer(_3N, 0.0f);

  for(unsigned int i = 0; i < _3N; i++) {
    hipMemcpy(&buffer[0], &blockEigenvectors[_3N + i], _3N, hipMemcpyDeviceToHost);
    for(unsigned int j = 0; j < _3N; j++) {
       blockVectors[j][i] = buffer[j];
    }
  } 
}

void FBMCuda::getProjectionMatrix(std::vector<std::vector<double > >& projMatrix) const {
  projMatrix.resize(_3N, std::vector<double>(m, 0.0));
  vector<float> buffer(_3N, 0.0f);

  for(unsigned int i = 0; i < m; i++) {
    hipMemcpy(&buffer[0], &E[_3N + i], _3N, hipMemcpyDeviceToHost);
    for(unsigned int j = 0; j < _3N; j++) {
       projMatrix[j][i] = buffer[j];
    }
  }
}

void FBMCuda::getHE(std::vector<std::vector<double > >& HEout) const {
  HEout.resize(_3N, std::vector<double>(m, 0.0));
  vector<float> buffer(_3N, 0.0f);

  for(unsigned int i = 0; i < m; i++) {
    hipMemcpy(&buffer[0], &HE[_3N + i], _3N, hipMemcpyDeviceToHost);
    for(unsigned int j = 0; j < _3N; j++) {
       HEout[j][i] = buffer[j];
    }
  } 
}

void FBMCuda::getCoarseGrainedHessian(std::vector<std::vector<double > >& coarseGrainedHessian) const {
  coarseGrainedHessian.resize(m, std::vector<double>(m, 0.0));
  vector<float> buffer(m, 0.0f);

  for(unsigned int i = 0; i < m; i++) {
    hipMemcpy(&buffer[0], &S[m + i], m, hipMemcpyDeviceToHost);
    for(unsigned int j = 0; j < m; j++) {
       coarseGrainedHessian[j][i] = buffer[j];
    }
  } 
}


FBMCuda::FBMCuda(Context &c, Context &bC, FBMParameters &p) : FBMAbstract(c, bC, p) { 
   // These will give us access to GPU pointers
   data = reinterpret_cast<CudaPlatform::PlatformData*>(getContextImpl(c).getPlatformData());
   blockData = reinterpret_cast<CudaPlatform::PlatformData*>(getContextImpl(bC).getPlatformData());
   
   // Set up the masses...
   System &system = c.getSystem();
   int _N = context.getState(State::Positions).getPositions().size() / 3;
   float* tmpmass = new float[_N];
   for (int i = 0; i < _N; i++)
      tmpmass[i] = system.getParticleMass(i);
   hipMalloc( (void**) &masses, _N*sizeof(float));
   hipMemcpy( masses, tmpmass, _N*sizeof(float), hipMemcpyHostToDevice);
   delete tmpmass;
}

void FBMCuda::makeBlocksAndThreads(int count) {
   if( count <= MAXTHREADSPERBLOCK) {
      numBlocks = 1;
      numThreads = count;
   } else {
       numBlocks = count / MAXTHREADSPERBLOCK + 1;
       numThreads = MAXTHREADSPERBLOCK;
   }
}

void FBMCuda::formBlocks() {
   // Make the blocks and put on GPU
   int block_start = 0;
   largestBlockSize = 0;
   vector<int> blocks;
   vector<int> blocksize;
   vector<int> hessiannum;
   vector<int> hessiansize;
   for( int i = 0; i < params.residue_sizes.size(); i++ ) {
      if( i % params.res_per_block == 0 ) {
         blocks.push_back( block_start );
	 hessiannum.push_back( block_start*block_start );
	 if (hessiannum.size() != 1)
	    hessiansize.push_back(hessiannum[hessiannum.size()-1]);
      }
      block_start += params.residue_sizes[i];
    }
   hessiansize.push_back(_3N-hessiannum[hessiannum.size()-1]);
  
  cout << "block sizes " << blocks[0] << " ";
   for( int i = 1; i < blocks.size(); i++ ) {
   cout << blocks[i] << " ";
	int block_size = blocks[i] - blocks[i - 1];
	blocksize.push_back( block_size );
	if( block_size > largestBlockSize ) {
	   largestBlockSize = block_size;
	}
   }
   cout << endl;
   numHessBlocks = blocks.size();
   largestBlockSize *= 3;
   cout << "Allocating memory" << endl;
   hipMalloc( (void**) &blocknums, blocks.size()*sizeof(int));
   hipMalloc( (void**) &blocksizes, blocks.size()*sizeof(int));
   hipMalloc( (void**) &hessiannums, blocks.size()*sizeof(int));
   hipMalloc( (void**) &hessiansizes, blocks.size()*sizeof(int));
   hipMemcpy(blocknums, &blocks[0], blocks.size()*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(blocksizes, &blocksize[0], blocks.size()*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(hessiannums, &hessiannum[0], blocks.size()*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(hessiansizes, &hessiansize[0], blocks.size()*sizeof(int), hipMemcpyHostToDevice);


   // Allocate a block Hessian on the GPU
   State state = context.getState( State::Positions | State::Forces );
   _3N = state.getPositions().size();
   int _N = _3N / 3;
   //hipMalloc( (void**) &blockHessian, _3N*_3N*sizeof(float));
   // The total number of elements in a linear block Hessian will be the sum of the squares of the block sizes
   int numelements = 0;
   for (int i = 0; i < blocksize.size(); i++)
      numelements += blocksize[i]*blocksize[i];
   hipMalloc( (void**) &blockHessian, numelements*sizeof(float));

   // Temporary buffers to hold positions and forces
   float* pos1;
   float* force1;
   hipMalloc( (void**) &pos1, _3N*sizeof(float));
   hipMalloc( (void**) &force1, _3N*sizeof(float));

   // GPU Thread organization...
   makeBlocksAndThreads(_N);

   cout << "Computing hessian" << endl;

   float4* positionArray;
   float4 myfloats;
   myfloats.x = 1.0;
   myfloats.y = 0.0;
   myfloats.z = 0.0;
   hipMalloc( (void**) &positionArray, _N * sizeof(float4));
   vector<float4> buffer(_N, myfloats);
   hipMemcpy(positionArray, &buffer[0], _N * sizeof(float4), hipMemcpyHostToDevice);

   blockContext.getState(State::Positions | State::Forces).getForces();

   ContextImpl impl = getContextImpl(blockContext);

   cout << "got Impl" << endl;

   impl.getTime();

   cout << "called impl positions" << endl;

   CudaPlatform::PlatformData* blockData = (CudaPlatform::PlatformData*) impl.getPlatformData();

   if(!blockData)
   cout << "Block data is NULL" << endl;
   else
   cout << "Block data is not NULL!" << endl;

   vector<Vec3> positionsCheck(_N);
   impl.getPositions(positionsCheck);

   cout << "Got positions" << endl;

   cout << positionsCheck[0][0] << " " << positionsCheck[0][1] << endl;

   vector<Vec3> try4 = blockContext.getState(State::Positions).getPositions();

   cout << try4[0][0] << " " << try4[0][1] << endl;
   
   if(blockData->gpu)
   cout << "GPU data not NULL!";
   else
   cout << "GPU data is NULL!";

   if(blockData->gpu->psPosq4)
   cout << "CUDAStream is not NULL!";
   else
   cout << "CUDAStream is NULL!";

   blockData->gpu->psPosq4->Upload();

   float4* positions = blockData->gpu->psPosq4->_pDevData;   

   // Populate the Hessian
   numelements = 0;
   for( unsigned int i = 0; i < largestBlockSize; i++ ) {
   cout << "current dof " << i << endl;      
      // Perturb in the (+) direction
      // After this, pos1 will hold the old positions and the positions pointer
      // has been directly perturbed
      perturbPositions<<<numBlocks, numThreads>>>(pos1, positions, params.blockDelta, blocknums, blocks.size(), i, _N);     
      cout << " Calling forces" << endl;
      // Compute forces
      blockContext.getState( State::Forces ).getForces();
      
      // Save first force vector
      //copyFromOpenMM<<<numBlocks, numThreads>>>(force1, &((*(data->gpu->psForce4))[0].w), _3N);
      //blockcopyFromOpenMM<<<numBlocks, numThreads>>>(force1, &((*(blockData->gpu->psPosq4))[0].w), blocknums, blocks.size(), i, _N);
      
      // Copy back positions
      //blockcopyToOpenMM<<<numBlocks, numThreads>>>(&((*(blockData->gpu->psPosq4))[0].w), pos1, blocknums, blocks.size(), i, _N);

      // Perturb in the the (-) direction 
      //perturbPositions<<<numBlocks, numThreads>>>(pos1, &((*(blockData->gpu->psPosq4))[0].w), -(params.blockDelta), blocknums, blocks.size(), i, _N);     

      // Compute forces
      //blockContext.getState( State::Forces ).getForces();

      // Allocate GPU memory for the Hessian
      // TMC THIS HAS A BUG
      // You have to determine that last parameter (starting spot) inside the function by passing block sizes.
      // It is not determined by degree of freedom
      // So I believe the second call is right, not the first
      //makeBlockHessian<<<numBlocks, numThreads>>>(blockHessian, force1, &((*(blockData->gpu->psForce4))[0].w), masses, params.blockDelta, blocknums, blocks.size(), i, _N, numelements); 
      //makeBlockHessian<<<numBlocks, numThreads>>>(blockHessian, force1, &((*(blockData->gpu->psForce4))[0].w), masses, params.blockDelta, blocknums, blocksizes, blocks.size(), i, _N); 
 
      numelements += blocksize[i]*blocksize[i];
   }

   cout << "computed hessian" << endl;

   // Symmetrize the Hessian
   makeBlocksAndThreads(_3N*_3N);
   symmetrize1D<<<numBlocks, numThreads>>>(blockHessian, _3N);
}


void FBMCuda::diagonalizeBlocks() {
   // The QR code expects the blocks to be placed in a linear array
   // Note: It also assumes that the number of matricies to diagonalize will be 
   // smaller than MAXBLOCKSPERTHREAD
   // I think this will mostly be the case anyway...
   float* tmp;
   hipMalloc( (void**) &blockEigenvalues, _3N*sizeof(float) );
   hipMalloc( (void**) &tmp, _3N*_3N*sizeof(float) );
   hipMalloc( (void**) &blockEigenvectors, _3N*sizeof(float));
   block_QR<<<1, numHessBlocks>>>(numHessBlocks, blockHessian, hessiannums, hessiansizes, tmp, hessiannums, 1, 1e-8);

   // Transfer elements from the diagonal of the block Hessian to the block Eigenvalues
   // blockEigenvectors should be good
   makeBlocksAndThreads(_3N);
   makeEigenvalues<<<numBlocks,numThreads>>>(blockEigenvalues, blockEigenvectors, blockHessian, tmp, hessiannums, blocknums, blocksizes, _3N);
   
   // Sort eigenvectors within each block
   makeBlocksAndThreads(numHessBlocks);
   blockEigSort<<<numBlocks, numThreads>>>(blockEigenvalues, blockEigenvectors, blocknums, hessiansizes);

   // Compute geometric degrees of freedom
   float* norms;
   float** poscenter;
   hipMalloc( (void**) &norms, numHessBlocks*sizeof(float) );
   hipMalloc( (void**) &poscenter, numHessBlocks*3*sizeof(float) );
   
   computeNormsAndCenter<<<numBlocks, numThreads>>>(norms, poscenter, masses, &((*(blockData->gpu->psPosq4))[0].w), blocknums, blocksizes);
   
   float*** Qi_gdof;
   hipMalloc( (void**) &Qi_gdof, numHessBlocks*largestBlockSize*largestBlockSize);
   geometricDOF<<<numBlocks, numThreads>>>(Qi_gdof, &((*(blockData->gpu->psPosq4))[0].w), masses, blocknums, blocksizes, norms, poscenter); 
  
   orthogonalize23<<<numHessBlocks, CONSERVEDDEGREESOFFREEDOM-4>>>(Qi_gdof, blocksizes);
   orthogonalize<<<numBlocks, numThreads>>>(blockEigenvectors, Qi_gdof, CONSERVEDDEGREESOFFREEDOM, blocksizes, blocknums);
}


void FBMCuda::formProjectionMatrix() {
   // Sort eigenvalues
   int oddcount = _3N/2;
   int evencount;
   if (_3N % 2 == 0) evencount = oddcount + 1;
   else evencount = oddcount;
   
   for (int i = 0; i < ceil(_3N/2); i++) {
      makeBlocksAndThreads(evencount);
      oddEvenEigSort<<<numBlocks, numThreads>>>(blockEigenvalues, blockEigenvectors);
      makeBlocksAndThreads(oddcount);
      oddEvenEigSort<<<numBlocks, numThreads>>>(blockEigenvalues, blockEigenvectors, 1);
   }

   // Copy the eigenvalues back and compute m (no way to parallelize that I can see)
   float* tmpeig = new float[_3N];
   hipMemcpy(tmpeig, blockEigenvalues, _3N*sizeof(float), hipMemcpyDeviceToHost);
   int max_eigs = params.bdof * numHessBlocks;
   float cutEigen = blockEigenvalues[max_eigs];
   vector<int> indices;
   int m = 0;
   for (int i = 0; i < _3N; i++)
   {
      if (blockEigenvalues[i] < cutEigen)
         indices.push_back(i);
      m++;
   }

   // Now we form the m X n matrix
   int* index;
   makeBlocksAndThreads(m*_3N);
   hipMalloc( (void**) &Et, m*_3N*sizeof(float));
   hipMalloc( (void**) &E, _3N*m*sizeof(float));
   hipMalloc( (void**) &index, indices.size()*sizeof(int));
   hipMemcpy(index, &indices[0], indices.size()*sizeof(int), hipMemcpyHostToDevice);
   makeProjection<<<numBlocks, numThreads>>>(Et, E, blockEigenvectors, index, _3N);
}

void FBMCuda::computeHE() {
   // Temporary buffers to hold positions and forces
   float* pos1;
   float*force1;
   hipMalloc( (void**) &pos1, _3N*sizeof(float));
   hipMalloc( (void**) &force1, _3N*sizeof(float));

   for (int k = 0; k < m; k++) {
       // Peturb positions, +
       makeBlocksAndThreads(_3N);

       // Perturb in the positive direction
       perturbByE<<<numBlocks, numThreads>>>(pos1, &((*(data->gpu->psPosq4))[0].w), params.sDelta, E, masses, k, _3N);

       // Calculate Forces
       context.getState( State::Forces ).getForces();
      
       // Save first force vector
       copyFromOpenMM<<<numBlocks, numThreads>>>(force1, &((*(data->gpu->psForce4))[0].w), _3N);
       
       // Copy back positions
       copyToOpenMM<<<numBlocks, numThreads>>>(&((*(data->gpu->psPosq4))[0].w), pos1, _3N);

       // Perturb in the the (-) direction 
       perturbByE<<<numBlocks, numThreads>>>(pos1, &((*(data->gpu->psPosq4))[0].w), -params.sDelta, E, masses, k, _3N);
       
       // Calculate Forces
       context.getState( State::Forces ).getForces();
      
       // Make HE
       hipMalloc( (void**) &HE, _3N*m*sizeof(float));
       makeHE<<<numBlocks, numThreads>>>(HE, force1, &((*(data->gpu->psForce4))[0].w), masses, params.sDelta, k, _3N);

       // Put back positions
       copyToOpenMM<<<numBlocks, numThreads>>>(&((*(data->gpu->psPosq4))[0].w), pos1, _3N);
   }
}

void FBMCuda::computeS() {
   makeBlocksAndThreads(m*m);

   hipMalloc( (void**) &S, m*m*sizeof(float));
   MatMulKernel<<<numBlocks, numThreads>>>(S, Et, HE, m, _3N);
   symmetrize2D<<<numBlocks, numThreads>>>(S, m);
}

void FBMCuda::diagonalizeS() {
   // Initialize Cula and check for errors
   culaStatus status = culaInitialize();
   if(status != culaNoError)
   {
     cout << culaGetStatusString(status) << endl;
   }

   // Temporary, for eigenvectors and eigenvalues of S
   // Cula populates the same array
   //float* tmpEigval = (float*) malloc(m*sizeof(float));
   //float** tmpEigvec = (float**) malloc(m*m*sizeof(float));

   // In the future we may want to find a way to directly access
   // Cula GPU arrays, rather than copy twice

   // Copy the eigenvalues into dS and eigenvectors into Q
   hipMalloc( (void**) &dS, m*sizeof(float) );
   //hipMemcpy(dS, tmpEigval, m*sizeof(float), hipMemcpyHostToDevice);

   hipMalloc( (float**) &Q, m*m*sizeof(float) );
   hipMemcpy(Q, S, m*m*sizeof(float), hipMemcpyDeviceToHost);
   //hipMemcpy(Q, tmpEigvec, m*m*sizeof(float), hipMemcpyHostToDevice);
   status = culaDeviceDsyev('V', 'U', m, (double*) &(Q[0][0]), m, (double*)dS);
}

void FBMCuda::computeModes(vector<double>& eigenvalues, vector<vector<Vec3> >& modes) {
   // A matrix multiply, but we have to copy back into formats the user expects
   makeBlocksAndThreads(_3N*m);
   float** U;
   hipMalloc( (float**) &U, _3N*m*sizeof(float) );
   MatMulKernel<<<numBlocks, numThreads>>>(U, E, Q, _3N, m);

   eigenvalues.resize(m);
   hipMemcpy(&(eigenvalues[0]), dS, m*sizeof(float), hipMemcpyDeviceToHost);
   
   float** myU = (float**) malloc(_3N*m*sizeof(float));
   hipMemcpy(myU, U, _3N*m*sizeof(float), hipMemcpyDeviceToHost);
  
   modes.resize(m);
   
   for (int i = 0; i < m; i++) 
      modes[i].resize(_3N/3);
   
   for (int i = 0; i < m; i++) {
      for (int j = 0; j < _3N; j += 3) {
         modes[i][j][0] = myU[i][j];
	 modes[i][j][1] = myU[i][j+1];
	 modes[i][j][2] = myU[i][j+2];
      }
   }
}
